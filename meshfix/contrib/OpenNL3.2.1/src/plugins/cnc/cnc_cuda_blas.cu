#include "hip/hip_runtime.h"
/*
 *  Copyright (c) 2004-2010, Bruno Levy
 *  All rights reserved.
 *
 *  CNC: Concurrent Number Cruncher, original code by Luc Buatois
 *  Copyright (C) 2008-2010 GOCAD/ASGA, INRIA/ALICE
 *
 *  Sparse matrix-vector multiplication (SpMV) CUDA kernels based on code
 *  by Nathan Bell and Michael Garland at NVIDIA.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions are met:
 *
 *  * Redistributions of source code must retain the above copyright notice,
 *  this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright notice,
 *  this list of conditions and the following disclaimer in the documentation
 *  and/or other materials provided with the distribution.
 *  * Neither the name of the ALICE Project-Team nor the names of its
 *  contributors may be used to endorse or promote products derived from this
 *  software without specific prior written permission.
 * 
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 *  ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 *  LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 *  CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 *  SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 *  INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 *  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 *  ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  If you modify this software, you should include a notice giving the
 *  name of the person performing the modification, the date of modification,
 *  and the reason for such modification.
 *
 *  Contact: Bruno Levy
 *
 *     levy@loria.fr
 *
 *     ALICE Project
 *     LORIA, INRIA Lorraine, 
 *     Campus Scientifique, BP 239
 *     54506 VANDOEUVRE LES NANCY CEDEX 
 *     FRANCE
 *
 */

// Header files
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "cnc_kernels.h"
#include "cnc_gpu_solver.h"
#include "cnc_texture.h"




// a convert fonctor to use with the transform algorithm
// when copying from double vector to float vector and vice-versa
template <typename Arg,typename Result>
struct convert : std::unary_function<Arg,Result> {
    Result operator() (const Arg x) {
        return  static_cast<Result>(x);
    }   
}; 


//---------------------------------------------------------------------------//
// kernel for matrix-product vector with a BCRS<float,2,2> matrix            //  
//---------------------------------------------------------------------------//
__global__ void CNCMat2x2VecMult4Kernel (
        float4 * matrix,
        unsigned int size_matrix,
        uint2 * rowptr,
        unsigned int size_rowptr,
        unsigned int * colind,
        unsigned int size_colind,
        const float2 * x,
        float2 * b,
        unsigned int size_vec ) {

	// Thread index
	const unsigned int index = large_grid_thread_id(void) ;

	if ( index<<1 < size_vec ) {
		uint2 rowptr_bounds = rowptr[index] ;
		float2 res ;
		res.x = res.y = 0.0f ;

		unsigned int ci = 0 ;
		float2 x_vec ;
		float4 mat_vec ;

		// for each block of the block_row, mult
		for ( int i=rowptr_bounds.x; i<rowptr_bounds.y; i++ ) { 
			ci = colind[i] ;
			mat_vec = matrix[i] ;
			#ifndef USE_TEXTURE
			x_vec = x[ci] ;
			#else
			x_vec = fetch_x(ci,x) ;
			#endif

			res.x += mat_vec.x*x_vec.x+mat_vec.y*x_vec.y ;
			res.y += mat_vec.z*x_vec.x+mat_vec.w*x_vec.y ;
		}
		b[index] = res ;
	}
}


//---------------------------------------------------------------------------//
// kernel for matrix-product vector with a BCRS<double,2,2> matrix           //  
//---------------------------------------------------------------------------//

__global__ void CNCMat2x2VecMult4Kernel (
        double2 * mat0,
        double2 *mat1 ,
        unsigned int size_matrix,
        uint2 * rowptr,
        unsigned int size_rowptr,
        unsigned int * colind,
        unsigned int size_colind,
        const double2 * x,
        double2 * b,
        unsigned int size_vec ) {

	// Thread index
	const unsigned int index = large_grid_thread_id(void); 

	if ( index<<1 < size_vec ) {

		uint2 rowptr_bounds = rowptr[index] ;
		double2 res ;
		res.x = res.y = 0.0;

		unsigned int ci = 0 ;
		double2 row0;
		double2 row1;
		double2 x_vec ;

		// for each block of the block_row, mult
		for ( int i=rowptr_bounds.x; i<rowptr_bounds.y; i++ ) { 

			row0 = mat0[i] ;
			row1 = mat1[i] ;
			ci = colind[i] ;
			#ifndef USE_TEXTURE
			x_vec = x[ci] ;
			#else
			x_vec = fetch_x(ci,x) ;
			#endif

			res.x += row0.x*x_vec.x+row0.y*x_vec.y ;
			res.y += row1.x*x_vec.x+row1.y*x_vec.y ;
		}
		b[index] = res ;
	}
}


//-------------------------------------------------------------------------------------//
// Main entry point for the CNC plugin solver : it solves Ax=b with                    //
// the matrix A is NL SparseMatrix Format                                              //
// the rhs b is double *                                                               //
// the vector solution x is double *                                                   //
// solvertype must belongs to { FLOAT_CRS | DOUBLE_CRS | FLOAT_BCRS2 | DOUBLE_BCRS2    //
//                              FLOAT_ELL | DOUBLE_ELL | FLOAT_HYB | DOUBLE_HYB }      //
//-------------------------------------------------------------------------------------//

NLuint cnc_solve_cg (
        NLSparseMatrix *A,
        NLdouble *b,
        NLdouble *x,
		NLuint nb_iter_max,
		NLdouble epsilon,
		NLint solver_type ) {
	
	//check cuda device support
	CNC_ASSERT ( CNCCheckDevice (), "No CUDA device found, please check the compatibility of your hardware with CUDA on NVIDIA's website" ) ;
	CNC_ASSERT ( A->n == A->m, "Non-square sparse matrix unsupported" ) ;
	CNC_ASSERT ( nlSolverIsCNC(solver_type), "Wrong solver type" ) ;
	CNC_ASSERT ( nb_iter_max>=1, "Wrong number if iterations" ) ;
	CNC_ASSERT ( epsilon>0.0, "Wrong convergence threshold" ) ;
        CNC_ASSERT ( CNCConfigureDevice (), "CUDA device could not be configured properly");

	NLuint block_size = 1;

    switch(solver_type) {
        case NL_CNC_FLOAT_CRS:
        case NL_CNC_DOUBLE_CRS:
        case NL_CNC_FLOAT_ELL:
        case NL_CNC_DOUBLE_ELL:
        case NL_CNC_FLOAT_HYB:
        case NL_CNC_DOUBLE_HYB: {
            block_size = 1;
        } break ;
        case NL_CNC_FLOAT_BCRS2:
        case NL_CNC_DOUBLE_BCRS2: {
            block_size = 2;
        } break ;
        default:
            nl_assert_not_reached ;
        
    }

    printf ( "############################################################\n" ) ;
	printf ( "Start Init CG Solver: data conversion, allocation and upload\n" ) ;
	printf ( "------------------------------------------------------------\n" ) ;
	printf ( "max iter: %d\ntolerance: %e\nblock_size: %dx%d\n",
			 nb_iter_max, epsilon, block_size, block_size ) ;
	printf ( "size vector: %d\n", A->n ) ;

	NLuint val_ret=static_cast<NLuint>(-1);
    

    switch(solver_type) {
        case NL_CNC_FLOAT_CRS:
        case NL_CNC_FLOAT_BCRS2: {
            printf("using SINGLE precision floating point.\n");
            CNCArray1d<float>  array_x( A->n );
            CNCArray1d<float>  array_b( A->n );	
	
	        std::transform(x,x+A->n,array_x.data(),convert<double,float>());
	        std::transform(b,b+A->n,array_b.data(),convert<double,float>());
	
	        if ( block_size == 1 ) {
                CNCSparseMatrixCRS<float> smcrs ;
		        convert_matrix<float> ( A, smcrs, false ) ;
		        val_ret = solve_cg_internal<CNCSparseMatrixCRS<float>, float> ( 
                    smcrs, array_b, array_x, nb_iter_max, epsilon ) ; 
	        } else if ( block_size == 2 ) {
		        CNCSparseMatrixBCRS<float,2,2> smbcrs2x2 ;
		        convert_matrix<float, 2, 2> ( A, smbcrs2x2 ) ;
		        val_ret = solve_cg_internal<CNCSparseMatrixBCRS<float,2,2> ,float > (  
			        smbcrs2x2, array_b, array_x, nb_iter_max, epsilon ) ; 
	        } else {
		        printf ( "Wrong Block size\n" ) ; // you should never reach this point...
	        }
		
	        std::transform(array_x.data(),array_x.data()+A->n,x,convert<float,double>());
	        std::transform(array_b.data(),array_b.data()+A->n,b,convert<float,double>());
        } break ;
        case NL_CNC_FLOAT_ELL:{
            printf("using SINGLE precision floating point.\n");
            CNCArray1d<float>  array_x( A->n );
            CNCArray1d<float>  array_b( A->n );	
	
	        std::transform(x,x+A->n,array_x.data(),convert<double,float>());
	        std::transform(b,b+A->n,array_b.data(),convert<double,float>());
	
            CNCSparseMatrixELL<float> smell ;
		    convert_matrix<float> ( A, smell) ;
		    val_ret = solve_cg_internal<CNCSparseMatrixELL<float>, float> ( 
		        smell, array_b, array_x,  nb_iter_max, epsilon ) ; 
	        std::transform(array_x.data(),array_x.data()+A->n,x,convert<float,double>());
	        std::transform(array_b.data(),array_b.data()+A->n,b,convert<float,double>());

        } break ;
        case NL_CNC_FLOAT_HYB:{
            printf("using SINGLE precision floating point.\n");
            CNCArray1d<float>  array_x( A->n );
            CNCArray1d<float>  array_b( A->n );	
	
	        std::transform(x,x+A->n,array_x.data(),convert<double,float>());
	        std::transform(b,b+A->n,array_b.data(),convert<double,float>());
	
            CNCSparseMatrixHYB<float> smhyb ;
		    convert_matrix<float> ( A, smhyb) ;
		    val_ret = solve_cg_internal<CNCSparseMatrixHYB<float>, float> ( 
		        smhyb, array_b, array_x,  nb_iter_max, epsilon ) ; 
	        
		    std::transform(array_x.data(),array_x.data()+A->n,x,convert<float,double>());
	        std::transform(array_b.data(),array_b.data()+A->n,b,convert<float,double>());

        } break ;
        case NL_CNC_DOUBLE_ELL: {
            CNC_ASSERT ( CNCCheckDeviceDoubleSupport() , "No support for double precision floating point on the CUDA device found" ) ;
            printf("Nice ! using DOUBLE precision floating point.\n");
            CNCArray1d<double>  array_x( A->n );
            CNCArray1d<double>  array_b( A->n );	
	
	        std::copy(x,x+A->n,array_x.data());
	        std::copy(b,b+A->n,array_b.data());
	
		    CNCSparseMatrixELL<double> smell ;
	    	convert_matrix<double> ( A, smell) ;
		    val_ret = solve_cg_internal<CNCSparseMatrixELL<double>, double> ( 
		        smell, array_b, array_x,  nb_iter_max, epsilon ) ; 
	
	        std::copy(array_x.data(),array_x.data()+A->n,x);
	        std::copy(array_b.data(),array_b.data()+A->n,b);

        } break ;
	    case NL_CNC_DOUBLE_HYB: {
            CNC_ASSERT ( CNCCheckDeviceDoubleSupport() , "No support for double precision floating point on the CUDA device found" ) ;
            printf("Nice ! using DOUBLE precision floating point.\n");
            CNCArray1d<double>  array_x( A->n );
            CNCArray1d<double>  array_b( A->n );	
	
	        std::copy(x,x+A->n,array_x.data());
	        std::copy(b,b+A->n,array_b.data());
	
	        CNCSparseMatrixHYB<double> smhyb ;
		    convert_matrix<double> ( A, smhyb ) ;
		    val_ret = solve_cg_internal<CNCSparseMatrixHYB<double>, double> ( 
	            smhyb, array_b, array_x,  nb_iter_max, epsilon ) ; 
	
	        std::copy(array_x.data(),array_x.data()+A->n,x);
	        std::copy(array_b.data(),array_b.data()+A->n,b);

        } break ;
        case NL_CNC_DOUBLE_CRS:
        case NL_CNC_DOUBLE_BCRS2: {
            CNC_ASSERT ( CNCCheckDeviceDoubleSupport() , "No support for double precision floating point on the CUDA device found" ) ;
        
            printf("Nice ! using DOUBLE precision floating point.\n");
            CNCArray1d<double>  array_x( A->n );
            CNCArray1d<double>  array_b( A->n );	
	
	        std::copy(x,x+A->n,array_x.data());
	        std::copy(b,b+A->n,array_b.data());
	
	        if ( block_size == 1 ) {
		        CNCSparseMatrixCRS<double> smcrs ;
		        convert_matrix<double> ( A, smcrs, false ) ;
		        val_ret = solve_cg_internal<CNCSparseMatrixCRS<double>, double> ( 
			        smcrs, array_b, array_x,  nb_iter_max, epsilon ) ; 
            } else if ( block_size == 2 ) {
		        CNCSparseMatrixBCRS<double,2,2> smbcrs2x2 ;
		        convert_matrix<double, 2, 2> ( A, smbcrs2x2 ) ;
		        val_ret = solve_cg_internal<CNCSparseMatrixBCRS<double,2,2> ,double> ( 
			        smbcrs2x2, array_b, array_x, nb_iter_max, epsilon ) ; 
	        } else {
		        printf ( "Wrong Block size\n" ) ; // you should never reach this point...
	        }
	        std::copy(array_x.data(),array_x.data()+A->n,x);
	        std::copy(array_b.data(),array_b.data()+A->n,b);
        } break ;
        default:
            nl_assert_not_reached ;
    }

	return val_ret;
}


